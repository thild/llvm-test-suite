
#include <hip/hip_runtime.h>
__global__ void kernel() {
  // Verify that *Idx/*Dim can be assigned to uint3/dim3.
  uint3 thread_idx = threadIdx;
  uint3 block_idx = blockIdx;
  dim3 block_dim = blockDim;
  dim3 grid_dim = gridDim;

  // And that they can be converted to uint3/dim3
  dim3 thread_idx_dim = threadIdx;
  dim3 block_idx_dim = blockIdx;
  uint3 block_dim_uint = blockDim;
  uint3 grid_dim_uint = gridDim;
}

int main(int argc, char* argv[]) {
  kernel<<<2, 2>>>();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
